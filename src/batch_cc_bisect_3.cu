#include "hip/hip_runtime.h"
/**********************************************************************
*  batch_cc_two_stage.cu                                              *
*                                                                     *
*  Two–stage batch collision checker                                   *
*    Stage-1 :  prune (mid-point test, 32 envs per block)             *
*    Stage-2 :  full discretised edge check on surviving pairs        *
**********************************************************************/

// this version checks 3 points during the prune stage
// (0.0, 0.5, 0.75) and uses a compacted list of pairs for the fine stage


// #pragma once

#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>

#include "src/collision/environment.hh"
#include "src/collision/factory.hh"
#include "src/Planners.hh"
#include "src/pRRTC_settings.hh"
#include "src/utils.cuh"
#include "batch_cc.hh"

#include <cassert>
#include <chrono>
#include <iostream>
#include <numeric>     // std::accumulate

namespace batch_cc
{
/*-----------------------------------------------------------*
 |  0.  helpers                                              |
 *-----------------------------------------------------------*/
inline __host__ __device__ constexpr int div_up(int a, int b)
{ return (a + b - 1) / b; }

/* (edge, env) pair used after compaction */
struct WorkPair { int edge; int env; };

/* one-byte flag per pair written by the prune kernel        *
 *  0 = keep for stage-2                                     *
 *  1 = already in collision (discard)                       */
using Flag = uint8_t;

inline void setup_environment_on_device(ppln::collision::Environment<float> *&d_env, 
    const ppln::collision::Environment<float> &h_env) {
    // First allocate the environment struct
    hipMalloc(&d_env, sizeof(ppln::collision::Environment<float>));

    // Initialize struct to zeros first
    hipMemset(d_env, 0, sizeof(ppln::collision::Environment<float>));

    // Handle each primitive type separately
    if (h_env.num_spheres > 0) {
    // Allocate and copy spheres array
    ppln::collision::Sphere<float> *d_spheres;
    hipMalloc(&d_spheres, sizeof(ppln::collision::Sphere<float>) * h_env.num_spheres);
    hipMemcpy(d_spheres, h_env.spheres, 
    sizeof(ppln::collision::Sphere<float>) * h_env.num_spheres, 
    hipMemcpyHostToDevice);

    // Update the struct fields directly
    hipMemcpy(&(d_env->spheres), &d_spheres, sizeof(ppln::collision::Sphere<float>*), 
    hipMemcpyHostToDevice);
    hipMemcpy(&(d_env->num_spheres), &h_env.num_spheres, sizeof(unsigned int), 
    hipMemcpyHostToDevice);
    }

    if (h_env.num_capsules > 0) {
    ppln::collision::Capsule<float> *d_capsules;
    hipMalloc(&d_capsules, sizeof(ppln::collision::Capsule<float>) * h_env.num_capsules);
    hipMemcpy(d_capsules, h_env.capsules,
    sizeof(ppln::collision::Capsule<float>) * h_env.num_capsules,
    hipMemcpyHostToDevice);

    hipMemcpy(&(d_env->capsules), &d_capsules, sizeof(ppln::collision::Capsule<float>*),
    hipMemcpyHostToDevice);
    hipMemcpy(&(d_env->num_capsules), &h_env.num_capsules, sizeof(unsigned int),
    hipMemcpyHostToDevice);
    }

    // Repeat for each primitive type...
    if (h_env.num_z_aligned_capsules > 0) {
    ppln::collision::Capsule<float> *d_z_capsules;
    hipMalloc(&d_z_capsules, sizeof(ppln::collision::Capsule<float>) * h_env.num_z_aligned_capsules);
    hipMemcpy(d_z_capsules, h_env.z_aligned_capsules,
    sizeof(ppln::collision::Capsule<float>) * h_env.num_z_aligned_capsules,
    hipMemcpyHostToDevice);

    hipMemcpy(&(d_env->z_aligned_capsules), &d_z_capsules, sizeof(ppln::collision::Capsule<float>*),
    hipMemcpyHostToDevice);
    hipMemcpy(&(d_env->num_z_aligned_capsules), &h_env.num_z_aligned_capsules, sizeof(unsigned int),
    hipMemcpyHostToDevice);
    }

    if (h_env.num_cylinders > 0) {
    ppln::collision::Cylinder<float> *d_cylinders;
    hipMalloc(&d_cylinders, sizeof(ppln::collision::Cylinder<float>) * h_env.num_cylinders);
    hipMemcpy(d_cylinders, h_env.cylinders,
    sizeof(ppln::collision::Cylinder<float>) * h_env.num_cylinders,
    hipMemcpyHostToDevice);

    hipMemcpy(&(d_env->cylinders), &d_cylinders, sizeof(ppln::collision::Cylinder<float>*),
    hipMemcpyHostToDevice);
    hipMemcpy(&(d_env->num_cylinders), &h_env.num_cylinders, sizeof(unsigned int),
    hipMemcpyHostToDevice);
    }

    if (h_env.num_cuboids > 0) {
    ppln::collision::Cuboid<float> *d_cuboids;
    hipMalloc(&d_cuboids, sizeof(ppln::collision::Cuboid<float>) * h_env.num_cuboids);
    hipMemcpy(d_cuboids, h_env.cuboids,
    sizeof(ppln::collision::Cuboid<float>) * h_env.num_cuboids,
    hipMemcpyHostToDevice);

    hipMemcpy(&(d_env->cuboids), &d_cuboids, sizeof(ppln::collision::Cuboid<float>*),
    hipMemcpyHostToDevice);
    hipMemcpy(&(d_env->num_cuboids), &h_env.num_cuboids, sizeof(unsigned int),
    hipMemcpyHostToDevice);
    }

    if (h_env.num_z_aligned_cuboids > 0) {
    ppln::collision::Cuboid<float> *d_z_cuboids;
    hipMalloc(&d_z_cuboids, sizeof(ppln::collision::Cuboid<float>) * h_env.num_z_aligned_cuboids);
    hipMemcpy(d_z_cuboids, h_env.z_aligned_cuboids,
    sizeof(ppln::collision::Cuboid<float>) * h_env.num_z_aligned_cuboids,
    hipMemcpyHostToDevice);

    hipMemcpy(&(d_env->z_aligned_cuboids), &d_z_cuboids, sizeof(ppln::collision::Cuboid<float>*),
    hipMemcpyHostToDevice);
    hipMemcpy(&(d_env->num_z_aligned_cuboids), &h_env.num_z_aligned_cuboids, sizeof(unsigned int),
    hipMemcpyHostToDevice);
    }
}

inline void cleanup_environment_on_device(ppln::collision::Environment<float> *d_env, 
    const ppln::collision::Environment<float> &h_env) {
    // Get the pointers from device struct before freeing
    ppln::collision::Sphere<float> *d_spheres = nullptr;
    ppln::collision::Capsule<float> *d_capsules = nullptr;
    ppln::collision::Capsule<float> *d_z_capsules = nullptr;
    ppln::collision::Cylinder<float> *d_cylinders = nullptr;
    ppln::collision::Cuboid<float> *d_cuboids = nullptr;
    ppln::collision::Cuboid<float> *d_z_cuboids = nullptr;

    // Copy each pointer from device memory
    if (h_env.num_spheres > 0) {
    hipMemcpy(&d_spheres, &(d_env->spheres), sizeof(ppln::collision::Sphere<float>*), hipMemcpyDeviceToHost);
    hipFree(d_spheres);
    }

    if (h_env.num_capsules > 0) {
    hipMemcpy(&d_capsules, &(d_env->capsules), sizeof(ppln::collision::Capsule<float>*), hipMemcpyDeviceToHost);
    hipFree(d_capsules);
    }

    if (h_env.num_z_aligned_capsules > 0) {
    hipMemcpy(&d_z_capsules, &(d_env->z_aligned_capsules), sizeof(ppln::collision::Capsule<float>*), hipMemcpyDeviceToHost);
    hipFree(d_z_capsules);
    }

    if (h_env.num_cylinders > 0) {
    hipMemcpy(&d_cylinders, &(d_env->cylinders), sizeof(ppln::collision::Cylinder<float>*), hipMemcpyDeviceToHost);
    hipFree(d_cylinders);
    }

    if (h_env.num_cuboids > 0) {
    hipMemcpy(&d_cuboids, &(d_env->cuboids), sizeof(ppln::collision::Cuboid<float>*), hipMemcpyDeviceToHost);
    hipFree(d_cuboids);
    }

    if (h_env.num_z_aligned_cuboids > 0) {
    hipMemcpy(&d_z_cuboids, &(d_env->z_aligned_cuboids), sizeof(ppln::collision::Cuboid<float>*), hipMemcpyDeviceToHost);
    hipFree(d_z_cuboids);
    }

    // Finally free the environment struct itself
    hipFree(d_env);
}


__device__ int write_index = 0;
/*-----------------------------------------------------------*
 |  1.  Stage-1 prune kernel                                 |
 *-----------------------------------------------------------*/
template <typename Robot>
__global__ void prune_kernel(
        ppln::collision::Environment<float>** envs,
        float* edges[2][Robot::dimension],
        int num_envs,
        int num_edges,
        bool* cc_result_full,
        WorkPair* work_pairs,
        float pct_along_edge
)
{
    constexpr int dim = Robot::dimension;

    /* grid layout -------------------------------------------------- *
     *  blockIdx.x  -> edge id                                        *
     *  blockIdx.y  -> stripe of 32 environments                      *
     *  threadIdx.x -> 0‥31, one environment inside the stripe        */
    const int edge_idx = blockIdx.x;
    const int env_idx  = blockIdx.y * 32 + threadIdx.x;

    if (edge_idx >= num_edges || env_idx >= num_envs) return;

    /* fetch edge endpoints (SoA) into registers */
    float start[dim], delta[dim], q0[dim], q1[dim], q2[dim];

    #pragma unroll
    for (int d = 0; d < dim; ++d) {
        start[d] = edges[0][d][edge_idx];
        delta[d] = edges[1][d][edge_idx] - start[d];
        q0[d] = start[d] + delta[d] * 0.0f;
        q1[d] = start[d] + delta[d] * 0.5f;
        q2[d] = start[d] + delta[d] * 0.75f;
    }

    /* midpoint collision test */
    bool coll0 = not ppln::collision::fkcc<Robot>(q0, envs[env_idx], /*lane=*/0);
    bool coll1 = not ppln::collision::fkcc<Robot>(q1, envs[env_idx], /*lane=*/0);
    bool coll2 = not ppln::collision::fkcc<Robot>(q2, envs[env_idx], /*lane=*/0);

    /* write flag (1 = collided already -> DISCARD) */
    // flag[edge_idx * num_envs + env_idx] = static_cast<Flag>(coll);
    bool coll = coll0 || coll1 || coll2;
    cc_result_full[edge_idx * num_envs + env_idx] = coll;
    
    if (!coll) {
        int idx = atomicAdd(&write_index, 1);
        work_pairs[idx].edge = edge_idx;
        work_pairs[idx].env = env_idx;
    }   
}

/*-----------------------------------------------------------*
 |  2.  Stage-2 fine kernel (mostly your old one)            |
 *-----------------------------------------------------------*/
template <typename Robot>
__global__ void fine_kernel(
        ppln::collision::Environment<float>** envs,
        float* edges[2][Robot::dimension],
        const WorkPair* work,         /* compact list           */
        int num_pairs,                /* <= gridDim.x           */
        bool* cc_result_full,         /* same shape as before   */
        int num_envs,                 /* to index flat array    */
        int resolution)
{
    constexpr int dim = Robot::dimension;
    const int tid = threadIdx.x;
    const int pair_idx = blockIdx.x;

    if (pair_idx >= num_pairs) return;

    /* map block -> (edge, env) */
    const int edge_idx = work[pair_idx].edge;
    const int env_idx  = work[pair_idx].env;

    ppln::collision::Environment<float>* env = envs[env_idx];

    /* shared memory for this block */
    __shared__ float edge_start[dim];
    __shared__ float edge_end  [dim];
    __shared__ float delta     [dim];
    __shared__ bool  local_cc_result;
    __shared__ int   n;

    /* load endpoints ------------------------------------------------*/
    if (tid < dim) {
        edge_start[tid] = edges[0][tid][edge_idx];
        edge_end  [tid] = edges[1][tid][edge_idx];
    }
    __syncthreads();

    /* discretisation count per lane -------------------------------- */
    if (tid == 0) {
        float dist = sqrt(device_utils::sq_l2_dist(edge_start, edge_end, dim));
        n = max(ceil((dist / (float) blockDim.x) * resolution), 1.0f);
        local_cc_result = false;
    }
    __syncthreads();

    if (tid < dim)
        delta[tid] = (edge_end[tid] - edge_start[tid]) / (float)(blockDim.x * n);
    __syncthreads();

    /* first configuration checked by each lane */
    float cfg[dim];
    #pragma unroll
    for (int d = 0; d < dim; ++d)
        cfg[d] = edge_start[d] + delta[d] * (tid * n);

    /* loop over n samples per lane ----------------------------------*/
    for (int i = 0; i < n; ++i) {
        bool in_collision = not ppln::collision::fkcc<Robot>(cfg, env, tid);
        local_cc_result = __any_sync(0xffffffff, in_collision);
        if (local_cc_result) break;

        #pragma unroll
        for (int d = 0; d < dim; ++d) cfg[d] += delta[d];
    }

    /* write final result */
    if (tid == 0)
        cc_result_full[edge_idx * num_envs + env_idx] = local_cc_result;
}




/*-----------------------------------------------------------*
 |  3.  Host-side entry – two-stage pipeline                 |
 *-----------------------------------------------------------*/
template <typename Robot>
void batch_cc(std::vector<ppln::collision::Environment<float>>& h_envs,
              std::vector<std::array<typename Robot::Configuration,2>>& edges,
              int resolution,
              std::vector<bool>& results)
{
    /* ---------- device-side environments -------------------------- */
    const int num_envs  = (int)h_envs.size();
    const int num_edges = (int)edges.size();

    std::vector<ppln::collision::Environment<float>*> d_envs(h_envs.size());
    for (size_t i = 0; i < h_envs.size(); ++i)
        setup_environment_on_device(d_envs[i], h_envs[i]);

    ppln::collision::Environment<float>** d_envs_ptr;
    hipMalloc(&d_envs_ptr, sizeof(ppln::collision::Environment<float>*) * num_envs);
    hipMemcpy(d_envs_ptr, d_envs.data(),
               sizeof(ppln::collision::Environment<float>*) * num_envs,
               hipMemcpyHostToDevice);

    /* ---------- SoA of edges -------------------------------------- */
    float* d_edges[2][Robot::dimension];
    for (int d = 0; d < Robot::dimension; ++d) {
        hipMalloc(&d_edges[0][d], sizeof(float) * num_edges);
        hipMalloc(&d_edges[1][d], sizeof(float) * num_edges);
        for (int e = 0; e < num_edges; ++e) {
            hipMemcpy(d_edges[0][d] + e, &edges[e][0][d],
                       sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(d_edges[1][d] + e, &edges[e][1][d],
                       sizeof(float), hipMemcpyHostToDevice);
        }
    }
    float* (*d_edges_ptr)[Robot::dimension];
    hipMalloc(&d_edges_ptr, sizeof(float*) * 2 * Robot::dimension);
    hipMemcpy(d_edges_ptr, d_edges,
               sizeof(float*) * 2 * Robot::dimension,
               hipMemcpyHostToDevice);
    
    // allocate final result array and work pairs array
    const int total_pairs = num_edges * num_envs;
    bool* d_cc_full;
    hipMalloc(&d_cc_full, sizeof(bool) * total_pairs);
    hipMemset(d_cc_full, 0, sizeof(bool) * total_pairs);
    WorkPair *d_work_pairs;
    hipMalloc(&d_work_pairs, sizeof(WorkPair) * total_pairs);


    const int stripe_cnt = div_up(num_envs, 32);
    dim3  block1(32);
    dim3  grid1(num_edges, stripe_cnt);

    auto start_time = std::chrono::steady_clock::now();
    auto prune_start_time = std::chrono::steady_clock::now();
    
    /* prune and compact 1 */
    prune_kernel<Robot><<<grid1, block1>>>(
        d_envs_ptr, d_edges_ptr,
        num_envs, num_edges,
        d_cc_full,
        d_work_pairs,
        0.5f
    );
    
    int num_remaining;
    hipMemcpyFromSymbol(&num_remaining, HIP_SYMBOL(write_index), sizeof(int), 0, hipMemcpyDeviceToHost);

    auto prune_time = get_elapsed_nanoseconds(prune_start_time);
    /* ---------- stage-2 fine kernel ------------------------------- */
    auto fine_start_time = std::chrono::steady_clock::now();
    if (num_remaining > 0) {
        int threads2 = 32;
        int blocks2  = num_remaining;

        fine_kernel<Robot><<<blocks2, threads2>>>(
            d_envs_ptr, d_edges_ptr,
            d_work_pairs,
            num_remaining,
            d_cc_full,
            num_envs,
            resolution);
    }

    hipDeviceSynchronize();
    auto fine_time = get_elapsed_nanoseconds(fine_start_time);
    auto total_time = get_elapsed_nanoseconds(start_time);

    cudaCheckError(hipGetLastError());


    std::cout << "Total time: " << total_time << " ns" << std::endl;
    std::cout << "Prune time: " << prune_time << " ns" << std::endl;
    std::cout << "Fine time: " << fine_time << " ns" << std::endl;
    std::cout << "Edges checked: " << total_pairs << std::endl;
    std::cout << "Edges remaining: " << num_remaining << std::endl;
    std::cout << "Edges pruned: " << total_pairs - num_remaining << std::endl;
    double throughput = total_pairs / (total_time / 1e9);
    std::cout << "Throughput: " << throughput << " edges/s" << std::endl;

    /* ---------- copy results back to host ------------------------- */
    // Create a temporary buffer for the results
    bool* h_cc_result = new bool[num_envs * num_edges];
    hipMemcpy(h_cc_result, d_cc_full, sizeof(bool) * num_envs * num_edges, hipMemcpyDeviceToHost);
    
    // Copy from temporary buffer to vector<bool>
    for (int i = 0; i < num_envs * num_edges; ++i) {
        results[i] = h_cc_result[i];
    }
    delete[] h_cc_result;

    /* ---------- clean up ----------------------------------------- */
    hipFree(d_cc_full);

    for (int d = 0; d < Robot::dimension; ++d) {
        hipFree(d_edges[0][d]);
        hipFree(d_edges[1][d]);
    }
    hipFree(d_edges_ptr);

    for (size_t i = 0; i < h_envs.size(); ++i)
        cleanup_environment_on_device(d_envs[i], h_envs[i]);
    hipFree(d_envs_ptr);
}

/*-----------------------------------------------------------*
 |  4.  explicit template instantiations                     |
 *-----------------------------------------------------------*/
template void batch_cc<typename ppln::robots::Panda>(
        std::vector<ppln::collision::Environment<float>>&,
        std::vector<std::array<typename ppln::robots::Panda::Configuration,2>>&,
        int,
        std::vector<bool>&);

template void batch_cc<typename ppln::robots::Fetch>(
        std::vector<ppln::collision::Environment<float>>&,
        std::vector<std::array<typename ppln::robots::Fetch::Configuration,2>>&,
        int,
        std::vector<bool>&);

template void batch_cc<typename ppln::robots::Baxter>(
        std::vector<ppln::collision::Environment<float>>&,
        std::vector<std::array<typename ppln::robots::Baxter::Configuration,2>>&,
        int,
        std::vector<bool>&);

} // namespace batch_cc
